#include "hip/hip_runtime.h"
/*   cuna_kernel.cu
 *   ==============   
 *   
 *   Implementation of the adjoint NFFT operation
 * 
 *   (c) John Hoffman 2016
 * 
 *   This file is part of CUNA
 *
 *   CUNA is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   CUNA is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with CUNA.  If not, see <http://www.gnu.org/licenses/>.
 */

// standard headers
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// CUDA headers
#include <hipfft/hipfft.h>

// local headers
#include "cuna.h"
#include "cuna_utils.h"
#include "cuna_filter.h"
#include "cuna_gridding.h"
#include "cufft_utils.h"

FILE *out;
char fname[200];

#ifdef DOUBLE_PRECISION
#define CUFFT_EXEC_CALL hipfftExecZ2Z
#define CUFFT_TRANSFORM_TYPE HIPFFT_Z2Z
#else
#define CUFFT_EXEC_CALL hipfftExecC2C
#define CUFFT_TRANSFORM_TYPE HIPFFT_C2C
#endif


__host__ void 
performGridding(plan *p) {

    int nblocks;
    nblocks = p->Ndata / BLOCK_SIZE;
    while (nblocks * BLOCK_SIZE < p->Ndata) nblocks++;
    
    LOG("about to do fast_gridding");
    // unequally spaced data -> equally spaced grid
    fast_gridding <<< nblocks, BLOCK_SIZE >>>
          ( p->g_f_data, p->g_f_grid, p->g_x_data, p->Ngrid,
            p->Ndata, p->fprops_device );

    if(p->flags & CALCULATE_WINDOW_FUNCTION) {
        LOG("about to do fast_gridding (WINDOW)");
        // unequally spaced data -> equally spaced grid
        fast_gridding <<< nblocks, BLOCK_SIZE >>>
              ( NULL, p->g_f_grid_win, p->g_x_data,
                p->Ngrid, p->Ndata, p->fprops_device );
    }

    // prints gridded data + window (if asked)
    if (p->flags & OUTPUT_INTERMEDIATE) {
        LOG("writing gridded_data");
        out = fopen("gridded_data.dat", "w");
        printReal_d(p->g_f_grid, p->Ngrid, out);
        fclose(out);
        if (p->flags & CALCULATE_WINDOW_FUNCTION) {
            LOG("writing gridded_data (WINDOW)");
            out = fopen("gridded_data_window.dat", "w");
            printReal_d(p->g_f_grid_win, p->Ngrid, out);
            fclose(out);
        }
    }
}

__host__ void 
transferGridResults(plan *p) {

    int nblocks;
    nblocks = p->Ngrid / BLOCK_SIZE;
    while(nblocks * BLOCK_SIZE < p->Ngrid) nblocks++; 

    LOG("converting g_f_grid to (Complex) g_f_hat");
    convertToComplex <<< nblocks, BLOCK_SIZE >>>
                   (p->g_f_grid, p->g_f_hat, p->Ngrid);

    if (p->flags & CALCULATE_WINDOW_FUNCTION) { 
        LOG("converting g_f_grid_win to (Complex) g_f_hat_win (WINDOW)");
        convertToComplex <<< nblocks, BLOCK_SIZE >>>
                   (p->g_f_grid_win, p->g_f_hat_win, p->Ngrid);
    }
}

__host__ void 
performFFTs(plan *p) {
    int nblocks;
    nblocks = p->Ngrid / BLOCK_SIZE;
    while(nblocks * BLOCK_SIZE < p->Ngrid) nblocks++; 

    LOG("calling hipfftPlan1d");
    
    // make plan
    hipfftHandle cuplan; 
    checkCufftError(
        hipfftPlan1d( &cuplan, p->Ngrid, CUFFT_TRANSFORM_TYPE, 1)
    );

    //LOG("synchronizing the device.");
    //checkCudaErrors(hipDeviceSynchronize());

    LOG("doing FFT of gridded data.");
    // FFT(gridded data)
    checkCufftError(
        CUFFT_EXEC_CALL( cuplan, p->g_f_hat, p->g_f_hat, HIPFFT_BACKWARD )
    );
    
    //LOG("synchronizing the device.");
    //checkCudaErrors(hipDeviceSynchronize());
    
    if (p->flags & OUTPUT_INTERMEDIATE) { 
        LOG("outputting raw fft of gridded data.");
        out = fopen("FFT_raw_f_hat.dat", "w");
        printComplex_d(p->g_f_hat, p->Ngrid, out);
        fclose(out);
    }

    if (p->flags & CALCULATE_WINDOW_FUNCTION) {
        
        LOG("doing FFT of gridded data. (WINDOW)");
        // FFT(gridded data)
        checkCufftError(
            CUFFT_EXEC_CALL( cuplan, p->g_f_hat_win,p->g_f_hat_win,
                             HIPFFT_BACKWARD)
        );

        if (p->flags & OUTPUT_INTERMEDIATE) { 
            LOG("outputting raw fft of gridded data. (WINDOW)");
            out = fopen("FFT_raw_f_hat_win.dat", "w");
            printComplex_d(p->g_f_hat_win, p->Ngrid, out);
            fclose(out);
        }

    }
    LOG("destroying cufft plan");
    hipfftDestroy(cuplan);
}

__host__ void 
normalizeResults(plan *p) {
    int nblocks;
    nblocks = p->Ngrid / BLOCK_SIZE;
    while(nblocks * BLOCK_SIZE < p->Ngrid) nblocks++; 

    LOG("Normalizing");
    // normalize (eq. 11 in Greengard & Lee 2004)
    normalize <<< nblocks, BLOCK_SIZE >>>
          ( p->g_f_hat, p->Ngrid, p->fprops_device );

    if(p->flags & CALCULATE_WINDOW_FUNCTION) {
        LOG("Normalizing (WINDOW)");
        // normalize (eq. 11 in Greengard & Lee 2004)
        normalize <<< nblocks, BLOCK_SIZE >>>
              ( p->g_f_hat_win, p->Ngrid, p->fprops_device );
    }
}


__host__ void 
copyResultsToCPU(plan *p) {
    LOG("Transferring data back to device");
    
    // Transfer back to device!
    checkCudaErrors(
        hipMemcpy( p->f_hat, p->g_f_hat, p->Ngrid * sizeof(Complex),
                    hipMemcpyDeviceToHost )
    );
    if(p->flags & CALCULATE_WINDOW_FUNCTION) {
        LOG("Transferring data back to device (WINDOW)");
    
        // Transfer back to device!
        checkCudaErrors(
            hipMemcpy( p->f_hat_win, p->g_f_hat_win,
                        p->Ngrid * sizeof(Complex), hipMemcpyDeviceToHost)
        );
    }
}

#define timeCommand(command)\
   if(p->flags | PRINT_TIMING) \
       start=clock(); \
   command;\
   if(p->flags | PRINT_TIMING) \
       fprintf(stderr, "  NFFT_ADJOINT.CU: %-20s : %.4e(s)\n", #command, seconds(clock() - start))

// computes the adjoint NFFT and stores this in plan->f_hat
__host__ void 
cunfft_adjoint_from_plan(plan *p) {
    clock_t start;
    timeCommand(performGridding(p));
    timeCommand(transferGridResults(p));
    timeCommand(performFFTs(p));
    timeCommand(normalizeResults(p));
    if ( !(p->flags | DONT_TRANSFER_TO_CPU)){
       timeCommand(copyResultsToCPU(p));
    }
}
