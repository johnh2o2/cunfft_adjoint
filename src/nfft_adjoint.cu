/* Implements the adjoint NFFT
 * 
 * (c) John Hoffman 2016
 * jah5@princeton.edu
 * 
 */

// local headers
#include "typedefs.h"
#include "filter.h"
#include "utils.h"
#include "adjoint_kernel.h"

// the usual headers
#include <stdlib.h>
#include <math.h>

// CUDA headers
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
//#include <helper_functions.h>
#include <hip/hip_runtime_api.h>



// computes the adjoint NFFT and stores this in plan->f_hat
__host__
void 
cuda_nfft_adjoint(
	plan 			*p

){

	unsigned int nblocks;
	nblocks = p->Ndata / BLOCK_SIZE;
	while(nblocks * BLOCK_SIZE < p->Ndata) nblocks++;
 
	// unequally spaced data -> equally spaced grid
	fast_gridding<<< nblocks, BLOCK_SIZE >>>(p->g_f_data, 
		p->g_f_hat, p->g_x_data, p->Ngrid, p->Ndata, p->fprops);

	// (same as above, but for the filter)
	fast_gridding<<< nblocks, BLOCK_SIZE >>>(NULL, 
		p->g_f_filter, p->g_x_data, p->Ngrid, p->Ndata, p->fprops);

	// make plan
	hipfftHandle cuplan;
	checkCudaErrors(
		hipfftPlan1d(&cuplan, p->Ngrid, HIPFFT_C2C, 1)
	);


	// FFT(gridded data)
	checkCudaErrors(
		hipfftExecC2C(cuplan, (hipfftComplex *)(p->g_f_hat), 
							(hipfftComplex *)(p->g_f_hat), HIPFFT_FORWARD )
	);

	// FFT(filter)
	checkCudaErrors(
		hipfftExecC2C(cuplan, (hipfftComplex *)(p->g_f_filter), 
							(hipfftComplex *)(p->g_f_filter), HIPFFT_FORWARD )
	);


	// FFT(gridded data) / FFT(filter)
	nblocks = p->Ngrid / BLOCK_SIZE;
	while(nblocks * BLOCK_SIZE < p->Ngrid) nblocks++;
	divide_by_spectral_window <<< nblocks, BLOCK_SIZE >>> (p->g_f_hat, p->g_f_filter, p->Ngrid);

	// normalize (eq. 11 in Greengard & Lee 2004)	
	normalize<<< nblocks, BLOCK_SIZE >>>(p->g_f_hat, p->Ngrid, p->fprops);

	// Transfer back to device!
	checkCudaErrors(hipMemcpy(p->f_hat, p->g_f_hat, p->Ngrid * sizeof(Complex),
		hipMemcpyDeviceToHost ));

	// Free plan memory.
	checkCudaErrors(hipfftDestroy(cuplan));
}

