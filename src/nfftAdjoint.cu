#include "hip/hip_runtime.h"
/*   nfft_adjoint.cu
 *   ===============   
 *   
 *   Implementation of the adjoint NFFT operation
 * 
 *   (c) John Hoffman 2016
 * 
 *   This file is part of cuNFFT_adjoint
 *
 *   cuNFFT_adjoint is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   cuNFFT_adjoint is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with cuNFFT_adjoint.  If not, see <http://www.gnu.org/licenses/>.
 */

// standard headers
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// CUDA headers
#include <hipfft/hipfft.h>

// local headers
#include "nfft_adjoint.h"

FILE *out;
char fname[200];

char * cufftParseError(hipfftResult_t r);
void checkCufftError(hipfftResult_t r);

#ifdef DOUBLE_PRECISION
#define CUFFT_EXEC_CALL hipfftExecZ2Z
#define CUFFT_TRANSFORM_TYPE HIPFFT_Z2Z

#else
#define CUFFT_EXEC_CALL hipfftExecC2C
#define CUFFT_TRANSFORM_TYPE HIPFFT_C2C

#endif

__global__
void
// shifts in k-space by -Ngrid/2 to be consistent with nfft3
convertToComplex(dTyp *a, Complex *c, int N){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		c[i].x = a[i];
		c[i].y = 0;
    }
}


__host__ void performGridding(plan *p) {

    int nblocks;
    nblocks = p->Ndata / BLOCK_SIZE;
    while (nblocks * BLOCK_SIZE < p->Ndata) nblocks++;
    
    LOG("about to do fast_gridding");
    // unequally spaced data -> equally spaced grid
    fast_gridding <<< nblocks, BLOCK_SIZE >>>(
                       p->g_f_data,
                       p->g_f_grid,
                       p->g_x_data,
                       p->Ngrid,
                       p->Ndata,
                       p->fprops_device
    );
    if(p->flags & CALCULATE_WINDOW_FUNCTION) {
        LOG("about to do fast_gridding (WINDOW)");
        // unequally spaced data -> equally spaced grid
        fast_gridding <<< nblocks, BLOCK_SIZE >>>(
                           NULL,
                           p->g_f_grid_win,
                           p->g_x_data,
                           p->Ngrid,
                           p->Ndata,
                           p->fprops_device
        );
    }

    if (p->flags & OUTPUT_INTERMEDIATE) {
        LOG("writing gridded_data");
        out = fopen("gridded_data.dat", "w");
        printReal_d(p->g_f_grid, p->Ngrid, out);
        fclose(out);
        if (p->flags & CALCULATE_WINDOW_FUNCTION) {
            LOG("writing gridded_data (WINDOW)");
            out = fopen("gridded_data_window.dat", "w");
            printReal_d(p->g_f_grid_win, p->Ngrid, out);
            fclose(out);
        }
    }
}

__host__ void transferGridResults(plan *p) {

    int nblocks;
    nblocks = p->Ngrid / BLOCK_SIZE;
    while(nblocks * BLOCK_SIZE < p->Ngrid) nblocks++; 

    LOG("converting g_f_grid to (Complex) g_f_hat");
    convertToComplex <<< nblocks, BLOCK_SIZE >>>
                   (p->g_f_grid, p->g_f_hat, p->Ngrid);

    if (p->flags & CALCULATE_WINDOW_FUNCTION) { 
        LOG("converting g_f_grid_win to (Complex) g_f_hat_win (WINDOW)");
        convertToComplex <<< nblocks, BLOCK_SIZE >>>
                   (p->g_f_grid_win, p->g_f_hat_win, p->Ngrid);
    }
}

__host__ void performFFTs(plan *p) {
    int nblocks;
    nblocks = p->Ngrid / BLOCK_SIZE;
    while(nblocks * BLOCK_SIZE < p->Ngrid) nblocks++; 

    LOG("calling hipfftPlan1d");
    
    // make plan
    hipfftHandle cuplan; 
    checkCufftError(
        hipfftPlan1d(
                   &cuplan,
                   p->Ngrid,
                   CUFFT_TRANSFORM_TYPE,
                   1
        )
    );
    //LOG("synchronizing the device.");
    //checkCudaErrors(hipDeviceSynchronize());

    LOG("doing FFT of gridded data.");
    // FFT(gridded data)
    checkCufftError(
        CUFFT_EXEC_CALL(  
                    cuplan,
                    p->g_f_hat,
                    p->g_f_hat,
                    HIPFFT_BACKWARD
        )
    );
    
    //LOG("synchronizing the device.");
    //checkCudaErrors(hipDeviceSynchronize());
    
    if (p->flags & OUTPUT_INTERMEDIATE) { 
        LOG("outputting raw fft of gridded data.");
        out = fopen("FFT_raw_f_hat.dat", "w");
        printComplex_d(p->g_f_hat, p->Ngrid, out);
        fclose(out);
    }

    if (p->flags & CALCULATE_WINDOW_FUNCTION) {
        
        LOG("doing FFT of gridded data. (WINDOW)");
        // FFT(gridded data)
        checkCufftError(
            CUFFT_EXEC_CALL(  
                        cuplan,
                        p->g_f_hat_win,
                        p->g_f_hat_win,
                        HIPFFT_BACKWARD
            )
        );

        if (p->flags & OUTPUT_INTERMEDIATE) { 
            LOG("outputting raw fft of gridded data. (WINDOW)");
            out = fopen("FFT_raw_f_hat_win.dat", "w");
            printComplex_d(p->g_f_hat_win, p->Ngrid, out);
            fclose(out);
        }

    }
    LOG("destroying cufft plan");
    hipfftDestroy(cuplan);
}

__host__ void normalizeResults(plan *p) {
    int nblocks;
    nblocks = p->Ngrid / BLOCK_SIZE;
    while(nblocks * BLOCK_SIZE < p->Ngrid) nblocks++; 

    LOG("Normalizing");
    // normalize (eq. 11 in Greengard & Lee 2004)
    normalize <<< nblocks, BLOCK_SIZE >>>(
            p->g_f_hat,
            p->Ngrid,
            p->fprops_device
    );

    if(p->flags & CALCULATE_WINDOW_FUNCTION) {
        LOG("Normalizing (WINDOW)");
        // normalize (eq. 11 in Greengard & Lee 2004)
        normalize <<< nblocks, BLOCK_SIZE >>>(
                p->g_f_hat_win,
                p->Ngrid,
                p->fprops_device
        );
    }
}


__host__ void copyResultsToCPU(plan *p) {
    LOG("Transferring data back to device");
    
    // Transfer back to device!
    checkCudaErrors(
        hipMemcpy(
            p->f_hat,
            p->g_f_hat,
            p->Ngrid * sizeof(Complex),
            hipMemcpyDeviceToHost
        )
    );
    if(p->flags & CALCULATE_WINDOW_FUNCTION) {
        LOG("Transferring data back to device (WINDOW)");
    
        // Transfer back to device!
        checkCudaErrors(
            hipMemcpy(
                p->f_hat_win,
                p->g_f_hat_win,
                p->Ngrid * sizeof(Complex),
                hipMemcpyDeviceToHost
            )
        );
    }
}

#define timeCommand(command)\
   if(p->flags | PRINT_TIMING) \
       start=clock(); \
   command;\
   if(p->flags | PRINT_TIMING) \
       fprintf(stderr, "  NFFT_ADJOINT.CU: %-20s : %.4e(s)\n", #command, seconds(clock() - start))

// computes the adjoint NFFT and stores this in plan->f_hat
__host__ void cuda_nfft_adjoint(plan *p) {
    clock_t start;
    timeCommand(performGridding(p));
    timeCommand(transferGridResults(p));
    timeCommand(performFFTs(p));
    timeCommand(normalizeResults(p));
    if ( !(p->flags | DONT_TRANSFER_TO_CPU)){
       timeCommand(copyResultsToCPU(p));
    }
}

char * cufftParseError(hipfftResult_t r){
    char *message = (char *) malloc( 100 * sizeof(char));
    switch(r){
        case HIPFFT_SUCCESS:
            sprintf(message, "The cuFFT operation was successful.");
            return message;
        case HIPFFT_INVALID_PLAN:
            sprintf(message, "cuFFT was passed an invalid plan handle.");
            return message;
        case HIPFFT_ALLOC_FAILED:
            sprintf(message, "cuFFT failed to allocate GPU or CPU memory.");
            return message;
        case HIPFFT_INVALID_TYPE:
            sprintf(message, "HIPFFT_INVALID_TYPE (no longer used)");
            return message;
        case HIPFFT_INVALID_VALUE:
            sprintf(message, "User specified an invalid pointer or parameter");
            return message;
        case HIPFFT_INTERNAL_ERROR:
            sprintf(message, "Driver or internal cuFFT library error.");
            return message;
        case HIPFFT_EXEC_FAILED:
            sprintf(message, "Failed to execute an FFT on the GPU.");
            return message;
        case HIPFFT_SETUP_FAILED:
            sprintf(message, "The cuFFT library failed to initialize.");
            return message;
        case HIPFFT_INVALID_SIZE:
            sprintf(message, "User specified an invalid transform size.");
            return message;
        case HIPFFT_UNALIGNED_DATA:
            sprintf(message, "HIPFFT_UNALIGNED_DATA (no longer used).");
            return message;
        case HIPFFT_INCOMPLETE_PARAMETER_LIST:
            sprintf(message, "Missing parameters in call.");
            return message;
        case HIPFFT_INVALID_DEVICE:
            sprintf(message, "Execution of a plan was on different GPU than plan creation. ");
            return message;
        case HIPFFT_PARSE_ERROR:
            sprintf(message, "Internal plan database error.");
            return message;
        case HIPFFT_NO_WORKSPACE:
            sprintf(message, "No workspace has been provided prior to plan execution.");
            return message;
        default:
            sprintf(message, "DONT UNDERSTAND THE CUFFT ERROR CODE!! %d", r);
            return message;
    }
}

void checkCufftError(hipfftResult_t r){
    if (r == HIPFFT_SUCCESS) return;

    fprintf(stderr, "cuFFT ERROR: %s\n", cufftParseError(r));
    exit(r);
}

/*
// used for debugging.
__host__
Complex *
testCufft(dTyp *x, dTyp *f, int N){

	// Allocate GPU memory
	Complex *d_x, *d_f;
	checkCudaErrors(
		hipMalloc((void **) &d_x, N * sizeof(Complex))
	);

	checkCudaErrors(
		hipMalloc((void **) &d_x, N * sizeof(Complex))
	);

	// Convert CPU data to complex
	Complex *xC = make_complex(x, N);
	Complex *fC = make_complex(f, N);

	// Copy data to GPU
	checkCudaErrors(
		hipMemcpy(d_x, xC, N * sizeof(Complex), hipMemcpyHostToDevice)
	);
	checkCudaErrors(
		hipMemcpy(d_f, fC, N * sizeof(Complex), hipMemcpyHostToDevice)
	);

	// Plan FFT
	hipfftHandle p;
	hipfftPlan1d( &p, N, HIPFFT_C2C, 1 );

	// Allocate space for FFT
	Complex *fft, *d_fft;

	// CPU
	fft = (Complex *)malloc(N * sizeof(Complex));

	// GPU
	checkCudaErrors(
		hipMalloc((void **)&d_fft, N * sizeof(Complex))
	);

	/////// DO FFT ///////
	hipfftExecC2C( p, d_f, d_fft, HIPFFT_BACKWARD );

	// Copy results back to CPU
	checkCudaErrors(
		hipMemcpy(fft, d_fft, N * sizeof(Complex), hipMemcpyDeviceToHost)
	);

	return fft;
}
*/
