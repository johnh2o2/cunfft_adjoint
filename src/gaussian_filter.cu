#include "hip/hip_runtime.h"
/*   gaussian_filter.cu
 *   ==================
 *   
 *   Implements the Gaussian filter
 * 
 *   (c) John Hoffman 2016
 * 
 *   This file is part of CUNA
 *
 *   CUNA is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   CUNA is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with CUNA.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <stdlib.h>
#include <stdio.h>

#include "cuna_filter.h"
#include "cuna_utils.h"

#ifdef DOUBLE_PRECISION
#define FILTER_RADIUS 12
#else
#define FILTER_RADIUS 6
#endif


__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const int Ngrid, 
				const int Ndata );


///////////////////////////////////////////////////////////////////////////////
// SET FILTER PROPERTIES + PRECOMPUTATIONS
__host__
void
generate_filter_properties(const dTyp *x, int n, int ng, filter_propertes **fprops_host, 
		filter_properties **fprops_device) {
	// Note: need two copies of the filter properties
	//       so that we can free the (E1, E2, E3) pointers
	//       because you can't do ([GPU]pointer)->(something)
	//       
	// [CPU]fprops_host   { [GPU]E1, E2, E3, [CPU]b, normfac, filter_radius }
	// [GPU]fprops_device { [GPU] ^,  ^,  ^, [GPU]b, normfac, filter_radius }  
	//    
	dTyp b, R;

	// nthreads = nblocks x BLOCK_SIZE
	int nblocks = (n + FILTER_RADIUS) / BLOCK_SIZE;

	// make sure nthreads >= data size + filter radius
	while (nblocks * BLOCK_SIZE < n + FILTER_RADIUS) nblocks++;

	// allocate host filter_properties
	*fprops_host = (filter_properties *)malloc(sizeof(filter_properties));

	// R                :  is the oversampling factor
	R = ((dTyp) ng) / n;

	// tau              :  is the characteristic length scale for the filter 
	//                     (not to be confused with the filter_radius)
	// NOTES:
	//     below was the expression I found in Greengard & Lee 2003; I think 
	//     they must have had a typo, since this tau is much too small.
	//
	//        tau = (1.0 / (p->Ndata * p->Ndata)) 
	// 			* (PI / (R* (R - 0.5))) * p->filter_radius;
	//tau = ((2 * R - 1)/ (2 * R)) * (PI / p->Ndata);
    b = 2 * R / (2 * R - 1) * (FILTER_RADIUS / PI);

	// set filter radius and shape parameter of (CPU) filter_properties
	(*fprops_host)->b             = b;
	(*fprops_host)->normfac       = sqrt(2 * PI); 
	(*fprops_host)->filter_radius = FILTER_RADIUS;

	
	// allocate (GPU) filter properties
	checkCudaErrors(
		hipMalloc((void **) fprops_device, sizeof(filter_properties))		
	);

	// allocate GPU memory for E1, E2, E3 of CPU filter_properties
	checkCudaErrors(
		hipMalloc((void **) &((*fprops_host)->E1), n * sizeof(dTyp))		
	);
	checkCudaErrors(
		hipMalloc((void **) &((*fprops_host)->E2), n * sizeof(dTyp))		
	);
	checkCudaErrors(
		hipMalloc((void **) &((*fprops_host)->E3), FILTER_RADIUS * sizeof(dTyp))		
	);

	// Copy filter properties to device
	checkCudaErrors(
		hipMemcpy(*fprops_device, *fprops_host, sizeof(filter_properties), 
						hipMemcpyHostToDevice )
	);

	// Precompute E1, E2, E3 on GPU
	set_gpu_filter_properties<<<nblocks, BLOCK_SIZE>>>(*fprops_device, x, ng, n);
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

///////////////////////////////////////////////////////////////////////////////
// SET UP FILTER FOR PLAN
__host__
void 
set_filter_properties(plan *p){
	LOG("in set_filter_properties");	

	dTyp b, R;

	// set plan filter radius
	p->filter_radius = FILTER_RADIUS;
	generate_filter_properties(p->g_x_data, p->Ndata, p->Ngrid, 
								&(p->fprops_host), &(p->fprops_device))
}


///////////////////////////////////////////////////////////////////////////////
// FREE GPU/CPU FILTER_PROPERTIES
__host__ void
free_filter_properties(filter_properties *d_fp, filter_properties *fp) {
	checkCudaErrors(hipFree(fp->E1));
	checkCudaErrors(hipFree(fp->E2));
	checkCudaErrors(hipFree(fp->E3));

	checkCudaErrors(hipFree(d_fp));
	free(fp);
}


///////////////////////////////////////////////////////////////////////////////
// Precomputation for filter (done on GPU)
__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const int Ngrid, 
				const int Ndata ){
	// index
	int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	if ( i < Ndata){
		
		// m = index of closest grid point to this data point
		int u = (int) (Ngrid * x[i] - f->filter_radius);
		
		// eps is the [0, 2pi] coordinate of the nearest gridpoint
		dTyp eps = Ngrid * x[i] - u;

		f->E1[i] = exp(- eps * eps /  f->b ) / sqrt(PI * f->b);
		f->E2[i] = exp(    2 * eps /  f->b ); 
	}
	else if ( i < Ndata + f->filter_radius){
		// E3 has just FILTER_RADIUS values
		int m = i - Ndata;
		f->E3[m] = exp( - m * m / f->b );
	}
	
}

///////////////////////////////////////////////////////////////////////////////
// Computes filter value for a given data index, grid index, and offset (m)
__device__
dTyp
filter( const int j_data, const int i_grid, 
				const int m , filter_properties *f){
	
	int mp;
	if (m < 0) mp = -m;
	else mp = m; 
	return f->E1[j_data] * pow(f->E2[j_data], m) * f->E3[mp];
}

///////////////////////////////////////////////////////////////////////////////
// Deconvolves filter from final result (analytically)

__global__
void
normalize(Complex *f_hat, int Ngrid, filter_properties *f){

	int k = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	if ( k < Ngrid ){
		dTyp K = ((dTyp) k) / ((dTyp) Ngrid);// - 0.5;
		dTyp fac = f->normfac * exp( K * K * f->b / 4. );
		f_hat[k].x *= fac;
		f_hat[k].y *= fac;
	}
}
