#include "hip/hip_runtime.h"
/* Implements the Gaussian filter for the NFFT
 *
 * (c) John Hoffman 2016
 * jah5@princeton.edu
 * 
 */

#include "filter.h"
#include "utils.h"
#include <stdlib.h>
#include <stdio.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>

#define FILTER_RADIUS 6

__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const unsigned int Ngrid, 
				const unsigned int Ndata );


__global__
void
set_gpu_filter_variables( filter_properties *f, dTyp tau, dTyp filter_radius){
	f->tau = tau;
	f->filter_radius = filter_radius;
}

// pre-computes values for the filter
__host__
void 
set_filter_properties(plan *p){
	LOG("in set_filter_properties");
	// nblocks x BLOCK_SIZE threads
	unsigned int nblocks = p->Ngrid / BLOCK_SIZE;

	// Ensures that we have enough threads!
	while (nblocks * BLOCK_SIZE < p->Ngrid) nblocks++;

	filter_properties *f;

	p->filter_radius = FILTER_RADIUS;

	LOG("hipMalloc f");
	checkCudaErrors(hipMalloc((void **) &f, sizeof(filter_properties)));

	// R                :  is the oversampling factor
	dTyp R = ((dTyp) p->Ngrid) / p->Ndata;

	// tau              :  is the characteristic length scale for the filter 
	//                     (not to be confused with the filter_radius)
	dTyp tau = (1.0 / (p->Ngrid * p->Ngrid)) * ( PI / (R* (R - 0.5)) ) * p->filter_radius;

	LOG("Setting tau and filter radius");
	// set tau and filter_radius (has to be done on GPU)
	set_gpu_filter_variables<<< 1, 1 >>>(f, tau, FILTER_RADIUS);

	LOG("hipMalloc f->E1");
	checkCudaErrors(hipMalloc((void **) &(f->E1), p->Ndata * sizeof(dTyp) ));
	LOG("hipMalloc f->E2");
	checkCudaErrors(hipMalloc((void **) &(f->E2), p->Ndata * sizeof(dTyp) ));
	LOG("hipMalloc f->E3");
	checkCudaErrors(hipMalloc((void **) &(f->E3), p->filter_radius * sizeof(dTyp) ));

	checkCudaErrors(hipDeviceSynchronize());

	LOG("calling setting_gpu_filter_properties");
	// Precompute E1, E2, E3 on GPU
	set_gpu_filter_properties<<<nblocks, BLOCK_SIZE>>>(f, p->x_data, p->Ngrid, p->Ndata);
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipDeviceSynchronize());
	

	LOG("setting plan->fprops to this filter_properties pointer");
	// Set plan's filter_properties pointer to this particular filter properties object
	p->fprops = f;

	checkCudaErrors(hipGetLastError());

}



/////////////////////////////////////////////
//  Uses GPU to precompute relevant values //
/////////////////////////////////////////////
__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const unsigned int Ngrid, 
				const unsigned int Ndata ){
	unsigned int i = get_index();
	if ( i < Ndata){
		unsigned int m = i * Ngrid / Ndata;
		dTyp eps = x[i] - 2 * PI * m / Ngrid;
		f->E1[i] = expf(- eps * eps / (4 * f->tau));
		f->E2[i] = expf( eps * PI / (Ngrid * f->tau)); 
	}
	if ( i < f->filter_radius){
		dTyp a = PI * PI * i * i / (Ngrid * Ngrid);
		f->E3[i] = expf( -a / f->tau);

	}
	
}

__device__
dTyp
filter( const unsigned int j_data, const unsigned int i_grid, 
				const int m , filter_properties *f){
	
	unsigned int mp;
	if (m < 0) mp = -m;
	else mp = m; 
	return f->E1[j_data] * powf(f->E2[j_data], m) * f->E3[mp];
}

__global__
void
normalize(Complex *f_hat, unsigned int Ngrid, filter_properties *f){

	unsigned int i = get_index();
	int k;
	if ( i < Ngrid ){
		k = i - Ngrid/2;
		f_hat[i].x *= sqrtf(PI/f->tau) * expf(k * k * f->tau);
	}
}
