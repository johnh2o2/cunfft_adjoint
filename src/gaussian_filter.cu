#include "hip/hip_runtime.h"
/* Implements the Gaussian filter for the NFFT
 *
 * (c) John Hoffman 2016
 * jah5@princeton.edu
 * 
 */

#include "filter.h"
#include "utils.h"
#include <stdlib.h>
#include <stdio.h>
//#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>

#define FILTER_RADIUS 6

__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const unsigned int Ngrid, 
				const unsigned int Ndata );

// pre-computes values for the filter
__host__
void 
set_filter_properties(plan *p){
	LOG("in set_filter_properties");
	// nblocks x BLOCK_SIZE threads
	unsigned int nblocks = p->Ngrid / BLOCK_SIZE;

	// Ensures that we have enough threads!
	while (nblocks * BLOCK_SIZE < p->Ngrid) nblocks++;

	LOG("CPU malloc for filter_properties pointer *f");
	// malloc memory for filter_properties (on GPU)
	filter_properties *f = (filter_properties *) malloc(sizeof(filter_properties));
	f->E1 = (dTyp *)malloc(p->Ndata * sizeof(dTyp));
	f->E2 = (dTyp *)malloc(p->Ndata * sizeof(dTyp));
	f->E3 = (dTyp *)malloc(p->filter_radius * sizeof(dTyp));

	f->filter_radius = FILTER_RADIUS;
	p->filter_radius = f->filter_radius;

	LOG("setting R and tau (CPU)");
	// R                :  is the oversampling factor
	dTyp R = ((dTyp) p->Ngrid) / p->Ndata;

	// tau              :  is the characteristic length scale for the filter 
	//                     (not to be confused with the filter_radius)
	f->tau = (1.0 / (p->Ngrid * p->Ngrid)) * ( PI / (R* (R - 0.5)) ) * p->filter_radius;

	//LOG("hipMalloc f");
	//checkCudaErrors(hipMalloc((void **) &f, sizeof(filter_properties) ));
	
	LOG("hipMalloc f->E1");
	checkCudaErrors(hipMalloc((void **) &(f->E1), p->Ndata * sizeof(dTyp) ));
	LOG("hipMalloc f->E2");
	checkCudaErrors(hipMalloc((void **) &(f->E2), p->Ndata * sizeof(dTyp) ));
	LOG("hipMalloc f->E3");
	checkCudaErrors(hipMalloc((void **) &(f->E3), p->filter_radius * sizeof(dTyp) ));


	filter_properties *d_f;
	LOG("hipMalloc d_f");
	checkCudaErrors(hipMalloc((void **) &d_f, sizeof(filter_properties)));

	LOG("hipMemcpy f -> d_f");
	checkCudaErrors(hipMemcpy(d_f, f, sizeof(filter_properties), hipMemcpyHostToDevice));


	LOG("callingf setting_gpu_filter_properties");
	// Precompute E1, E2, E3 on GPU
	set_gpu_filter_properties<<<nblocks, BLOCK_SIZE>>>(d_f, p->x_data, p->Ngrid, p->Ndata);

	LOG("setting plan->fprops to this filter_properties pointer");
	// Set plan's filter_properties pointer to this particular filter properties object
	p->fprops = d_f;

}



/////////////////////////////////////////////
//  Uses GPU to precompute relevant values //
/////////////////////////////////////////////
__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const unsigned int Ngrid, 
				const unsigned int Ndata ){
	size_t i = get_index();
	if ( i < Ndata){
		unsigned int m = i * Ngrid / Ndata;
		dTyp eps = x[i] - 2 * PI * m / Ngrid;
		f->E1[i] = expf(- eps * eps / (4 * f->tau));
		f->E2[i] = expf( eps * PI / (Ngrid * f->tau)); 
	}
	if ( i < f->filter_radius){
		dTyp a = PI * PI * i * i / (Ngrid * Ngrid);
		f->E3[i] = expf( -a / f->tau);
	}
	
}

__device__
dTyp
filter( const unsigned int j_data, const unsigned int i_grid, 
				const int m , filter_properties *f){
	
	unsigned int mp;
	if (m < 0) mp = -m;
	else mp = m; 
	return f->E1[j_data] * powf(f->E2[j_data], m) * f->E3[mp];
}

__global__
void
normalize(Complex *f_hat, unsigned int Ngrid, filter_properties *f){

	unsigned int i = get_index();
	int k;
	if ( i < Ngrid ){
		k = i - Ngrid/2;
		f_hat[i].x *= sqrtf(PI/f->tau) * expf(k * k * f->tau);
	}
}
