#include "hip/hip_runtime.h"
/*   gaussian_filter.cu
 *   ==================
 *   
 *   Implements the Gaussian filter
 * 
 *   (c) John Hoffman 2016
 * 
 *   This file is part of CUNA
 *
 *   CUNA is free software: you can redistribute it and/or modify
 *   it under the terms of the GNU General Public License as published by
 *   the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   CUNA is distributed in the hope that it will be useful,
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *   GNU General Public License for more details.
 *
 *   You should have received a copy of the GNU General Public License
 *   along with CUNA.  If not, see <http://www.gnu.org/licenses/>.
 */
#include <stdlib.h>
#include <stdio.h>

#include "cuna_filter.h"
#include "cuna_utils.h"

#ifdef DOUBLE_PRECISION
#define FILTER_RADIUS 12
#else
#define FILTER_RADIUS 6
#endif


__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const int Ngrid, 
				const int Ndata );

///////////////////////////////////////////////////////////////////////////////
// SET FILTER PROPERTIES + PRECOMPUTATIONS
__host__
void 
set_filter_properties(plan *p){

	// Note: need two copies of the filter properties
	//       so that we can free the (E1, E2, E3) pointers
	//       because you can't do ([GPU]pointer)->(something)
	//       
	// [CPU]p { [CPU]fprops_host   { [GPU]E1, E2, E3, [CPU]b, normfac, filter_radius },
	//          [GPU]fprops_device { [GPU] ^,  ^,  ^, [GPU]b, normfac, filter_radius }  
	//       }


	LOG("in set_filter_properties");
	
	dTyp b, R;

	// set plan filter radius
	p->filter_radius = FILTER_RADIUS;

	// nthreads = nblocks x BLOCK_SIZE
	int nblocks = (p->Ndata + p->filter_radius) / BLOCK_SIZE;

	// make sure nthreads >= data size + filter radius
	while (nblocks * BLOCK_SIZE < p->Ndata + p->filter_radius) nblocks++;

	// allocate host filter_properties
	LOG("malloc p->fprops_host");
	p->fprops_host = (filter_properties *)malloc(sizeof(filter_properties));

	// R                :  is the oversampling factor
	R = ((dTyp) p->Ngrid) / p->Ndata;

	// tau              :  is the characteristic length scale for the filter 
	//                     (not to be confused with the filter_radius)
	// NOTES:
	//     below was the expression I found in Greengard & Lee 2003; I think 
	//     they must have had a typo, since this tau is much too small.
	//
	//        tau = (1.0 / (p->Ndata * p->Ndata)) 
	// 			* (PI / (R* (R - 0.5))) * p->filter_radius;
	//tau = ((2 * R - 1)/ (2 * R)) * (PI / p->Ndata);
    b = 2 * R / (2 * R - 1) * (p->filter_radius / PI);

	LOG("setting p->fprops_host->(filter_radius, tau)");
	// set filter radius and shape parameter of (CPU) filter_properties
	p->fprops_host->b = b;
	p->fprops_host->normfac = sqrt(2 * PI); // * p->Ngrid;
	p->fprops_host->filter_radius = p->filter_radius;

	
	LOG("cuda malloc p->fprops_device");
	// allocate (GPU) filter properties
	checkCudaErrors(
		hipMalloc(
			(void **) &(p->fprops_device), 
			sizeof(filter_properties)
			)
		);

	
	LOG("hipMalloc p->fprops_host->E(1,2,3)");
	// allocate GPU memory for E1, E2, E3 of CPU filter_properties
	checkCudaErrors(
		hipMalloc(
			(void **) &(p->fprops_host->E1), 
			p->Ndata * sizeof(dTyp)
			)
		);
	checkCudaErrors(
		hipMalloc(
			(void **) &(p->fprops_host->E2), 
			p->Ndata * sizeof(dTyp)
			)
		);
	checkCudaErrors(
		hipMalloc(
			(void **) &(p->fprops_host->E3), 
			p->filter_radius * sizeof(dTyp)
			)
		);

	LOG("hipMemcpy p->fprops_host ==> p->fprops_device");
	// Copy filter properties to device
	checkCudaErrors(
		hipMemcpy(
			p->fprops_device, 
			p->fprops_host, 
			sizeof(filter_properties), 
			hipMemcpyHostToDevice 
			)
		);

	LOG("calling setting_gpu_filter_properties");
	// Precompute E1, E2, E3 on GPU
	set_gpu_filter_properties<<<nblocks, BLOCK_SIZE>>> (p->fprops_device, 
						p->g_x_data, p->Ngrid, p->Ndata);
	
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
	

}



///////////////////////////////////////////////////////////////////////////////
// Precomputation for filter (done on GPU)
__global__
void
set_gpu_filter_properties( filter_properties *f, dTyp *x, const int Ngrid, 
				const int Ndata ){
	// index
	int i = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	if ( i < Ndata){
		
		// m = index of closest grid point to this data point
		int u = (int) (Ngrid * x[i] - f->filter_radius);
		
		// eps is the [0, 2pi] coordinate of the nearest gridpoint
		dTyp eps = Ngrid * x[i] - u;

		f->E1[i] = exp(- eps * eps /  f->b ) / sqrt(PI * f->b);
		f->E2[i] = exp(    2 * eps /  f->b ); 
	}
	else if ( i < Ndata + f->filter_radius){
		// E3 has just FILTER_RADIUS values
		int m = i - Ndata;
		f->E3[m] = exp( - m * m / f->b );
	}
	
}

///////////////////////////////////////////////////////////////////////////////
// Computes filter value for a given data index, grid index, and offset (m)
__device__
dTyp
filter( const int j_data, const int i_grid, 
				const int m , filter_properties *f){
	
	int mp;
	if (m < 0) mp = -m;
	else mp = m; 
	return f->E1[j_data] * pow(f->E2[j_data], m) * f->E3[mp];
}

///////////////////////////////////////////////////////////////////////////////
// Deconvolves filter from final result (analytically)

__global__
void
normalize(Complex *f_hat, int Ngrid, filter_properties *f){

	int k = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	if ( k < Ngrid ){
		dTyp K = ((dTyp) k) / ((dTyp) Ngrid);// - 0.5;
		dTyp fac = f->normfac * exp( K * K * f->b / 4. );
		f_hat[k].x *= fac;
		f_hat[k].y *= fac;
	}
}
