#include "utils.h"
#include "filter.h"
#include <stdlib.h>



// Copies over a float array to Complex array
// TODO: Find a more efficient/sensible way to do this.
void copy_float_to_complex(dTyp *a, Complex *b, unsigned int N){
	for (unsigned int i = 0; i < N; i++){
		b[i].x = a[i];
		b[i].y = 0;
	}
}

void scale_x(dTyp *x, unsigned int size){
	// ensures that x \in [0, 2pi)

	float range = x[size-1] - x[0];
	for(unsigned int i = 0; i < size; i++){
		x[i]-=x[0];
		x[i]/=range;
		x[i] *= 2 * PI;
	}
}


void free_plan(plan *p){
	LOG("===== free_plan =====");
	LOG("free     p->f_hat");
	free(p->f_hat);
	LOG("free     p->x_data");
	free(p->x_data);
	LOG("free     p->f_data");
	free(p->f_data);

	LOG("hipFree p->fprops_host->E(1,2,3)");
	checkCudaErrors(hipFree(p->fprops_host->E1));
	checkCudaErrors(hipFree(p->fprops_host->E2));
	checkCudaErrors(hipFree(p->fprops_host->E3));

	LOG("free     p->fprops_host");
	free(p->fprops_host);

	LOG("hipFree p->fprops_device");
	checkCudaErrors(hipFree(p->fprops_device));

	LOG("hipFree p->g_f_hat");
	checkCudaErrors(hipFree(p->g_f_hat));

	LOG("hipFree p->g_f_filter");
	checkCudaErrors(hipFree(p->g_f_filter));

	LOG("hipFree p->g_f_data");
	checkCudaErrors(hipFree(p->g_f_data));

	LOG("hipFree p->g_x_data");
	checkCudaErrors(hipFree(p->g_x_data));

	LOG("free     p");
	free(p);

	LOG("=====================");
}

__host__
void 
init_plan(
	plan 			*p, 
	dTyp 			*f, 
	dTyp 			*x, 
	unsigned int 	Ndata, 
	unsigned int 	Ngrid

){
	LOG("in init_plan -- mallocing for CPU");
	p->Ndata = Ndata;
	p->Ngrid = Ngrid;
	p->x_data = (dTyp *)    malloc( Ndata * sizeof(dTyp));
	p->f_data = (dTyp *)    malloc( Ndata * sizeof(dTyp));
	p->f_hat  = (Complex *) malloc( Ngrid * sizeof(Complex));

	LOG("memcpy x and f to plan");
	memcpy(p->x_data, x, Ndata * sizeof(dTyp));
	memcpy(p->f_data, f, Ndata * sizeof(dTyp));

	// Allocate GPU memory
	LOG("hipMalloc -- p->g_f_data");
	checkCudaErrors(
		hipMalloc((void **) &(p->g_f_data), 
			p->Ndata * sizeof(Complex))
	);
	LOG("hipMalloc -- p->g_x_data");
	checkCudaErrors(
		hipMalloc((void **) &(p->g_x_data), 
			p->Ndata * sizeof(dTyp))
	);
	LOG("hipMalloc -- p->g_f_hat");
	checkCudaErrors(
		hipMalloc((void **) &(p->g_f_hat), 
			p->Ngrid * sizeof(Complex))
	);

	LOG("hipMalloc -- p->g_f_filter");
	checkCudaErrors(
		hipMalloc((void **) &(p->g_f_filter), 
			p->Ngrid * sizeof(Complex))
	);

	checkCudaErrors(hipDeviceSynchronize());

	LOG("copying f_data to f_data_complex");
	// "Cast" float array to Complex array
	Complex f_data_complex[p->Ndata];
	copy_float_to_complex(p->f_data, f_data_complex, p->Ndata);

	LOG("hipMemcpy f_data_complex ==> p->g_f_data");
	// Copy f_j -> GPU
	checkCudaErrors(
		hipMemcpy(p->g_f_data, f_data_complex, 
			p->Ndata * sizeof(float), hipMemcpyHostToDevice)
	);

	LOG("hipMemcpy p->x_data ==> p->g_x_data");
	// Copy x_j -> GPU
	checkCudaErrors(
		hipMemcpy(p->g_x_data, p->x_data, 
			p->Ndata * sizeof(float), hipMemcpyHostToDevice)
	);

	checkCudaErrors(hipDeviceSynchronize());

	LOG("done here, calling set_filter_properties");
	// copy filter information + perform 
	// precomputation
	set_filter_properties(p);

}
